#include "hip/hip_runtime.h"
#include "cudaKernel.h"
#include "stdio.h"

__device__ bool intersectSphere(Ray *ray, BoundingSphere *theSphere, Float3D viewCenter, float *t)
{
    const float EPS = 0.00001;
    float t0=0.0, t1=0.0, A=0.0, B=0.0, C=0.0, discrim=0.0;
    Float3D RoMinusSc = ray->Ro.minus(viewCenter);
    float fourAC = 0.0;

    A = ray->Rd.dot(ray->Rd);
    B = 2.0 * (ray->Rd.dot(RoMinusSc));
    C = RoMinusSc.dot(RoMinusSc) - theSphere->radiusSq;
    fourAC = (4*A*C);

    discrim = (B*B) - fourAC;

    if (discrim < EPS)
        return false;
    else
    {
        t0 = ((-B) - sqrt(discrim))/(2.0*A);
        t1 = ((-B) + sqrt(discrim))/(2.0*A);

        if (t0 < EPS)
        {
            if (t1 < EPS)
            {
                *t = 0.0;
                return false;
            }
            else
            {
                *t = t1;
                return true;
            }
        }
        else if (t1 < EPS)
        {
            *t = t0;
            return true;
        }
        else if (t0 < t1)
        {
            *t = t0;
            return true;
        }
        else
        {
            *t = t1;
            return true;
        }
    }
}

__device__ bool intersectTriangle(Ray *ray, Mesh *theObj, int v1, int v2, int v3, HitRecord *hrec, bool cull)
{
    Float3D verts[3] = {theObj->vertArray[v1], theObj->vertArray[v2], theObj->vertArray[v3]};
    Float3D edges[2];
    Float3D vnorms[3] = {theObj->viewNormArray[v1], theObj->viewNormArray[v2], theObj->viewNormArray[v3]};
    Float3D pvec, qvec, tvec;
    float det, inv_det;
    float EPSILON = 0.000001;

    edges[0] = verts[1].minus(verts[0]);
    edges[1] = verts[2].minus(verts[0]);
    pvec = ray->Rd.cross(edges[1]);
    det = edges[0].dot(pvec);
    if(cull)
    {
        if (det < EPSILON)
            return false;
        tvec = ray->Ro.minus(verts[0]);
        hrec->u = tvec.dot(pvec);
        if (hrec->u < 0.0 || hrec->u > det)
            return false;
        qvec = tvec.cross(edges[0]);
        hrec->v = ray->Rd.dot(qvec);
        if (hrec->v < 0.0 || hrec->u + hrec->v > det)
            return false;
        hrec->t = edges[1].dot(qvec);
        inv_det = 1.0/det;
        hrec->t *= inv_det;
        hrec->u *= inv_det;
        hrec->v *= inv_det;
    }
    else
    {
        if (det > -EPSILON && det < EPSILON)
            return false;
        inv_det = 1.0/det;
        tvec = ray->Ro.minus(verts[0]);
        hrec->u = tvec.dot(pvec) * inv_det;
        if (hrec->u < 0.0 || hrec->u > 1.0)
            return false;
        qvec = tvec.cross(edges[0]);
        hrec->v = ray->Rd.dot(qvec) * inv_det;
        if (hrec->v < 0.0 || hrec->u + hrec->v > 1.0)
            return false;
        if (det < -EPSILON)
            hrec->backfacing = true;
        else
            hrec->backfacing = false;
        hrec->t = edges[1].dot(qvec) * inv_det;
    }
    if (hrec->t < EPSILON)
        return false;
    else
    {
        hrec->intersectPoint = Float3D((ray->Ro.x + (ray->Rd.x * hrec->t)), (ray->Ro.y + (ray->Rd.y * hrec->t)), (ray->Ro.z + (ray->Rd.z * hrec->t)));
        float w = 1.0 - hrec->u - hrec->v;
        Float3D sumNorms(0.0, 0.0, 0.0);
        vnorms[0] = vnorms[0].sMult(w);
        vnorms[1] = vnorms[1].sMult(hrec->u);
        vnorms[2] = vnorms[2].sMult(hrec->v);
        sumNorms = vnorms[0].plus(vnorms[1].plus(vnorms[2]));
        hrec->normal = sumNorms;
        hrec->normal.unitize();
        return true;
    }
}

void cudaStart(Bitmap *bitmap, Mesh *objects, int numObjects, LightCuda *lights, int numLights, Options *options)
{
    int numRays;
    unsigned char *d_bitmap;
    unsigned char *h_bitmap;
    unsigned char *layers[options->maxRecursiveDepth + 1];
    Mesh *d_objects;
    Mesh *h_objects;
    LightCuda *d_lights;
    Ray *rays;
    Intersect *intersects;

    CHECK_ERROR(hipMalloc((void**)&d_bitmap, bitmap->width * bitmap->height * 3));
    h_bitmap = (unsigned char*)malloc(sizeof(unsigned char) * (bitmap->width * bitmap->height * 3));

    bitmap->data = d_bitmap;

    h_objects = (Mesh *)malloc(sizeof(Mesh) * numObjects);
    memcpy(h_objects, objects, sizeof(Mesh) * numObjects);

    for (int x = 0; x < numObjects; x++)
    {
        h_objects[x].surfaces = new Surface[h_objects[x].numSurfs];
        memcpy(h_objects[x].surfaces, objects[x].surfaces, sizeof(Surface) * objects[x].numSurfs);
        for (int y = 0; y < h_objects[x].numSurfs; y++)
        {
            CHECK_ERROR(hipMalloc((void**)&h_objects[x].surfaces[y].verts, sizeof(int) * h_objects[x].surfaces[y].numVerts));
            CHECK_ERROR(hipMemcpy(h_objects[x].surfaces[y].verts, objects[x].surfaces[y].verts, sizeof(int) * h_objects[x].surfaces[y].numVerts, hipMemcpyHostToDevice));
            delete [] objects[x].surfaces[y].verts;
            objects[x].surfaces[y].verts = h_objects[x].surfaces[y].verts;
            h_objects[x].surfaces[y].verts = NULL;
        }

        delete [] h_objects[x].surfaces;
        CHECK_ERROR(hipMalloc((void **)&h_objects[x].surfaces, sizeof(Surface) * h_objects[x].numSurfs));
        CHECK_ERROR(hipMemcpy(h_objects[x].surfaces, objects[x].surfaces, sizeof(Surface) * h_objects[x].numSurfs, hipMemcpyHostToDevice));
        CHECK_ERROR(hipMalloc((void **)&h_objects[x].materials, sizeof(Material) * h_objects[x].numMats));
        CHECK_ERROR(hipMemcpy(h_objects[x].materials, objects[x].materials, sizeof(Material) * h_objects[x].numMats, hipMemcpyHostToDevice));
        CHECK_ERROR(hipMalloc((void **)&h_objects[x].vertArray, sizeof(Float3D) * h_objects[x].numVerts));
        CHECK_ERROR(hipMemcpy(h_objects[x].vertArray, objects[x].vertArray, sizeof(Float3D) * h_objects[x].numVerts, hipMemcpyHostToDevice));
        CHECK_ERROR(hipMalloc((void **)&h_objects[x].viewNormArray, sizeof(Float3D) * h_objects[x].numVerts));
        CHECK_ERROR(hipMemcpy(h_objects[x].viewNormArray, objects[x].viewNormArray, sizeof(Float3D) * h_objects[x].numVerts, hipMemcpyHostToDevice));
    }

    CHECK_ERROR(hipMalloc((void**)&d_objects, sizeof(Mesh) * numObjects));
    CHECK_ERROR(hipMemcpy(d_objects, h_objects, sizeof(Mesh) * numObjects, hipMemcpyHostToDevice));

    CHECK_ERROR(hipMalloc((void**)&d_lights, sizeof(LightCuda) * numLights));
    CHECK_ERROR(hipMemcpy(d_lights, lights, sizeof(LightCuda) * numLights, hipMemcpyHostToDevice));

    numRays = bitmap->width * bitmap->height;
    CHECK_ERROR(hipMalloc((void**)&rays, sizeof(Ray) * numRays));

    CHECK_ERROR(hipMalloc((void**)&intersects, sizeof(Intersect) * numRays));

    for(int i = 0; i <= options->maxRecursiveDepth; i++)
    {
        CHECK_ERROR(hipMalloc((void**)&layers[i], sizeof(unsigned char) * (bitmap->width * bitmap->height * 4)));
        CHECK_ERROR(hipMemset(layers[i], 0, sizeof(unsigned char) * bitmap->width * bitmap->height * 4));
    }

    dim3 blocks((bitmap->width+15)/16, (bitmap->height+15)/16);
    dim3 threads(16, 16);
    //kernel<<<blocks, threads>>>(*bitmap, d_objects, numObjects, d_lights, numLights, *options);
    for(int pass = 0; pass <= options->maxRecursiveDepth; pass++)
    {
        baseKrnl<<<blocks, threads>>>(rays, numRays, *bitmap);
        intersectKrnl<<<blocks, threads>>>(rays, numRays, d_objects, numObjects, options->spheresOnly, intersects, options->cull);
        shadeKrnl<<<blocks, threads>>>(rays, numRays, intersects, layers[pass], d_lights, numLights, *options, pass == options->maxRecursiveDepth ? true : false);
        composeKrnl<<<blocks, threads>>>(*bitmap, layers[pass], pass == options->maxRecursiveDepth ? true : false);
    }

    CHECK_ERROR(hipMemcpy(h_bitmap, d_bitmap, bitmap->width * bitmap->height * 3, hipMemcpyDeviceToHost));

    CHECK_ERROR_FREE(hipFree(d_bitmap), &d_bitmap);

    for (int x = 0; x < numObjects; x++)
    {
        for (int y = 0; y < h_objects[x].numSurfs; y++)
        {
            CHECK_ERROR(hipFree(objects[x].surfaces[y].verts));
            objects[x].surfaces[y].verts = NULL;
        }
        CHECK_ERROR_FREE(hipFree(h_objects[x].surfaces), &h_objects[x].surfaces);
        CHECK_ERROR_FREE(hipFree(h_objects[x].materials), &h_objects[x].materials);
        CHECK_ERROR_FREE(hipFree(h_objects[x].vertArray), &h_objects[x].vertArray);
        CHECK_ERROR_FREE(hipFree(h_objects[x].viewNormArray), &h_objects[x].viewNormArray);
    }
    CHECK_ERROR_FREE(hipFree(d_objects), &d_objects);

    CHECK_ERROR_FREE(hipFree(d_lights), &d_lights);

    CHECK_ERROR_FREE(hipFree(rays), &rays);

    CHECK_ERROR_FREE(hipFree(intersects), &intersects);

    for(int i = 0; i <= options->maxRecursiveDepth; i++)
        CHECK_ERROR_FREE(hipFree(layers[i]), &layers[i]);

    bitmap->data = h_bitmap;

    free(h_objects);
}

void checkError(hipError_t error, const char *file, int line, void **nullObject)
{
    if (nullObject != NULL)
        nullObject = NULL;

    if (error != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(error), file, line);
        exit(EXIT_FAILURE);
    }
}

__global__ void baseKrnl(Ray *rays, int numRays, Bitmap bitmap)
{
    //Map from threadIdx & blockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    if (offset < (bitmap.width * bitmap.height))
    {
        Float3D point(bitmap.firstPixel);
        point.x += (offset % bitmap.width) * bitmap.pixelWidth;
        point.y += ((offset - x) / bitmap.width) * bitmap.pixelHeight;
        rays[offset] = Ray(point.getUnit(), Float3D(0.0, 0.0, 0.0), EYE);
    }
}

__global__ void intersectKrnl(Ray *rays, int numRays, Mesh *objects, int numObjects, bool spheresOnly, Intersect *intrs, bool cull)
{
    //Map from threadIdx & blockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    if (offset < numRays)
    {
        float t = 0.0;
        float intersectDist = 0.0;
        float minDist = 100000000.0;
        int minMatIndex = 0;
        bool minBackfacing = false;
        Mesh *minObj = NULL;
        Float3D minIntPt(0.0, 0.0, 0.0);
        Float3D minNormal(0.0, 0.0, 0.0);
        Float3D intersectPt(0.0, 0.0, 0.0);
        Float3D normal(0.0, 0.0, 0.0);
        Float3D origin(0.0, 0.0, 0.0);

        for (int obj = 0; obj < numObjects; obj++)
        {
            if (intersectSphere(&rays[offset], &(objects[obj].boundingSphere), objects[obj].viewCenter, &t))
            {
                if (abs(t) < 0.0001)
                    continue;
                if (spheresOnly)
                {
                    intersectPt = Float3D((rays[offset].Ro.x+(rays[offset].Rd.x*t)), (rays[offset].Ro.y+(rays[offset].Rd.y*t)), (rays[offset].Ro.z+(rays[offset].Rd.z*t)));
                    normal = (intersectPt.minus(objects[obj].viewCenter).sDiv(objects[obj].boundingSphere.radius));
                    normal.unitize();
                    intersectDist = origin.distanceTo(intersectPt);
                    if (intersectDist < minDist)
                    {
                        minDist = intersectDist;
                        minObj = &objects[obj];
                        minIntPt = Float3D(intersectPt);
                        minNormal = Float3D(normal);
                    }
                }
                else
                {
                    for (int surf = 0; surf < objects[obj].numSurfs; surf++)
                    {
                        for (int i =  0; i < (int)(objects[obj].surfaces[surf].numVerts / 3); i++)
                        {
                            HitRecord hrec;
                            if (intersectTriangle(&rays[offset], &objects[obj], objects[obj].surfaces[surf].verts[i*3], objects[obj].surfaces[surf].verts[(i*3)+1], objects[obj].surfaces[surf].verts[(i*3)+2], &hrec, false))
                            {
                                if (!(rays[offset].flags == EYE && hrec.backfacing && cull) || rays[offset].flags == REFLECT)
                                {
                                    intersectDist = rays[offset].Ro.distanceTo(hrec.intersectPoint);
                                    if (intersectDist < minDist)
                                    {
                                        minDist = intersectDist;
                                        minObj = &objects[obj];
                                        minIntPt = hrec.intersectPoint;
                                        minNormal = hrec.normal;
                                        minMatIndex = objects[obj].surfaces[surf].material;
                                        minBackfacing = hrec.backfacing;
                                    }
                                }
                            }
                        }
                    }
                }
            }
        }
        intrs[offset] = Intersect(minMatIndex, minBackfacing, minObj, minIntPt, minNormal);
    }
}

__global__ void shadeKrnl(Ray *rays, int numRays, Intersect *intrs, unsigned char *layer, LightCuda *lights, int numLights, Options options, bool finalPass)
{
    //Map from threadIdx & blockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    if (offset < numRays && intrs[offset].theObj != NULL)
    {
        int materialIndex = intrs[offset].materialIndex;
        Mesh *theObj = intrs[offset].theObj;
        FloatColor Ka(0.0, 0.0, 0.0, 1.0);
        FloatColor Kd(0.0, 0.0, 0.0, 1.0);
        FloatColor Ks(0.0, 0.0, 0.0, 1.0);
        FloatColor shadeColor(0.0, 0.0, 0.0, 1.0);
        FloatColor ambColor(0.0, 0.0, 0.0, 1.0);
        Float3D point = intrs[offset].point;
        Float3D trueNormal(0.0, 0.0, 0.0);
        Float3D inv_normal = intrs[offset].normal.sMult(-1.0);
        Float3D R(0.0, 0.0, 0.0);
        Float3D L(0.0, 0.0, 0.0);
        Float3D V(0.0, 0.0, 0.0);

        layer[offset*4 + 3] = (int) (theObj->materials[materialIndex].reflectivity.r * 255);

        Ka = theObj->materials[materialIndex].ka;
        Kd = theObj->materials[materialIndex].kd;
        Ks = theObj->materials[materialIndex].ks;

        ambColor.r = Ka.r * lights[0].ambient.r;
        ambColor.g = Ka.g * lights[0].ambient.g;
        ambColor.b = Ka.b * lights[0].ambient.b;

        shadeColor.plus(ambColor);
        V = Float3D(0.0, 0.0, 0.0).minus(point);
        V.unitize();

        if (rays[offset].flags == EYE && intrs[offset].backFacing && !options.cull)
            trueNormal = inv_normal;
        else
            trueNormal = intrs[offset].normal;

        LightCuda *curLight;
        for (int i = 0; i < numLights; i++)
        {
            curLight = &lights[i];

            L = curLight->viewPosition.minus(point);
            L.unitize();
            float LdotN = L.dot(trueNormal);
            LdotN = max(0.0, LdotN);
            FloatColor diffComponent(0.0, 0.0, 0.0, 1.0);
            if (LdotN > 0.0)
                diffComponent.plus(FloatColor(curLight->diffuse.r*Kd.r*LdotN, curLight->diffuse.g*Kd.g*LdotN, curLight->diffuse.b*Kd.b*LdotN, 1.0));
            shadeColor.plus(diffComponent);

            Float3D Pr = trueNormal.sMult(LdotN);
            Float3D sub = Pr.sMult(2.0);
            R = L.sMult(-1.0).plus(sub);
            R.unitize();
            float RdotV = R.dot(V);
            RdotV = max(0.0, RdotV);
            float cosPhiPower = 0.0;
            if (RdotV > 0.0)
                cosPhiPower = pow(RdotV, theObj->materials[materialIndex].shiny);
            FloatColor specComponent(curLight->specular.r*Ks.r*cosPhiPower, curLight->specular.g*Ks.g*cosPhiPower, curLight->specular.b*Ks.b*cosPhiPower, 1.0);
            shadeColor.plus(specComponent);
        }

        shadeColor.r = shadeColor.r < 0.0 ? 0.0 : (shadeColor.r > 1.0 ? 1.0 : shadeColor.r);
        shadeColor.g = shadeColor.g < 0.0 ? 0.0 : (shadeColor.g > 1.0 ? 1.0 : shadeColor.g);
        shadeColor.b = shadeColor.b < 0.0 ? 0.0 : (shadeColor.b > 1.0 ? 1.0 : shadeColor.b);

        if (finalPass)
        {
            layer[offset*4 + 0] = shadeColor.r * 255;
            layer[offset*4 + 1] = shadeColor.g * 255;
            layer[offset*4 + 2] = shadeColor.b * 255;
        }
    }
}

__global__ void composeKrnl(Bitmap bitmap, unsigned char *layer, bool finalPass)
{
    //Map from threadIdx & blockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    if (offset < (bitmap.width * bitmap.height))
    {
        if (finalPass)
        {
            bitmap.data[offset*3 + 0] = layer[offset*4 + 0];
            bitmap.data[offset*3 + 1] = layer[offset*4 + 1];
            bitmap.data[offset*3 + 2] = layer[offset*4 + 2];
        }
        else
        {

        }
    }

}
