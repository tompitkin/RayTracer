#include "hip/hip_runtime.h"
#include "cudaKernel.h"
#include "stdio.h"

/*__device__ int numObjects;
__device__ Mesh *objects;
__device__ int numLights;
__device__ LightCuda * lights;
__device__ Options *options;

__global__ void kernel(Bitmap bitmap, Mesh *d_objects, int d_numObjects, LightCuda *d_lights, int d_numLights, Options d_options)
{
    //Map from threadIdx & blockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    if (offset < (bitmap.width * bitmap.height))
    {
        numObjects = d_numObjects;
        objects = d_objects;
        numLights = d_numLights;
        lights = d_lights;
        options = &d_options;

        Float3D point(bitmap.firstPixel);
        point.x += (offset % bitmap.width) * bitmap.pixelWidth;
        point.y += ((offset - x) / bitmap.width) * bitmap.pixelHeight;
        Ray ray(point.getUnit(), Float3D(), EYE);

        DoubleColor rgb = trace(ray, 0);

        bitmap.data[offset*3 + 0] = (int) (rgb.r * 255);
        bitmap.data[offset*3 + 1] = (int) (rgb.g * 255);
        bitmap.data[offset*3 + 2] = (int) (rgb.b * 255);
    }
}

__device__ DoubleColor trace(Ray ray, int numRecurs)
{
    double t = 0.0;
    double intersectDist = 0.0;
    double minDist = 100000000.0;
    int minMatIndex = 0;
    bool minBackfacing = false;
    Mesh *minObj = NULL;
    Float3D minIntPt;
    Float3D minNormal;
    Float3D intersectPt;
    Float3D normal;
    Float3D origin;

    for (int obj = 0; obj < numObjects; obj++)
    {
        if (intersectSphere(ray, &objects[obj], &t))
        {
            if (abs(t) < 0.00001)
                continue;
            if (options->spheresOnly)
            {
                intersectPt = Float3D((ray.Ro.x+(ray.Rd.x*t)), (ray.Ro.y+(ray.Rd.y*t)), (ray.Ro.z+(ray.Rd.z*t)));
                normal = (intersectPt.minus(objects[obj].viewCenter).sDiv(objects[obj].boundingSphere.radius));
                normal.unitize();
                intersectDist = origin.distanceTo(intersectPt);
                if (intersectDist < minDist)
                {
                    minDist = intersectDist;
                    minObj = &objects[obj];
                    minIntPt = Float3D(intersectPt);
                    minNormal = Float3D(normal);
                }
            }
            else
            {
                for (int surf = 0; surf < objects[obj].numSurfs; surf++)
                {
                    for (int i =  0; i < (int)(objects[obj].surfaces[surf].numVerts / 3); i++)
                    {
                        HitRecord hrec;
                        if (intersectTriangle(&ray, &objects[obj], objects[obj].surfaces[surf].verts[i*3], objects[obj].surfaces[surf].verts[(i*3)+1], objects[obj].surfaces[surf].verts[(i*3)+2], &hrec, false))
                        {
                            if (!(ray.flags == EYE && hrec.backfacing && options->cull) || ray.flags == REFLECT || ray.flags == EXTERNAL_REFRACT)
                            {
                                intersectDist = ray.Ro.distanceTo(hrec.intersectPoint);
                                if (intersectDist < minDist)
                                {
                                    minDist = intersectDist;
                                    minObj = &objects[obj];
                                    minIntPt = hrec.intersectPoint;
                                    minNormal = hrec.normal;
                                    minMatIndex = objects[obj].surfaces[surf].material;
                                    minBackfacing = hrec.backfacing;
                                }
                            }
                        }
                    }
                }
            }
        }
    }
    if (minObj != NULL)
    {
        return shade(minObj, minIntPt, minNormal, minMatIndex, minBackfacing, ray, numRecurs);
    }
    else
    {
        return DoubleColor(0.0, 0.0, 0.0, 1.0);
    }
}

__device__ DoubleColor shade(Mesh *theObj, Float3D point, Float3D normal, int materialIndex, bool backFacing, Ray ray, int numRecurs)
{
    DoubleColor shadeColor;
    DoubleColor reflColor;
    DoubleColor refrColor;
    DoubleColor Ka;
    DoubleColor Kd;
    DoubleColor Ks;
    Float3D inv_normal = normal.sMult(-1.0);
    Float3D trueNormal;
    bool reflections;
    bool refractions;
    double reflectivity;
    double refractivity;

    if (ray.flags == INTERNAL_REFRACT)
        reflections = false;
    else
        reflections = options->reflections;
    refractions = options->refractions;
    reflectivity = theObj->materials[materialIndex].reflectivity.r;
    refractivity = theObj->materials[materialIndex].refractivity.r;

    Ka = theObj->materials[materialIndex].ka;
    Kd = theObj->materials[materialIndex].kd;
    Ks = theObj->materials[materialIndex].ks;

    Float3D R;
    Float3D L;
    Float3D V;

    shadeColor.plus(lights[0].ambient);

    V = Float3D(0.0, 0.0, 0.0).minus(point);
    V.unitize();

    if (ray.flags == EYE && backFacing && !options->cull)
        trueNormal = inv_normal;
    else if (ray.flags == INTERNAL_REFRACT && backFacing)
        trueNormal = inv_normal;
    else
        trueNormal = normal;

    LightCuda *curLight;
    for (int i = 0; i < numLights; i++)
    {
        bool obstructed = false;
        curLight = &lights[i];

        if(options->shadows)
        {
            Float3D Rd(curLight->viewPosition.minus(point));
            Rd.unitize();
            Ray shadowRay = Ray(Float3D(Rd), Float3D(point));
            if (traceLightRay(shadowRay))
                obstructed = true;
        }
        if (obstructed)
            continue;

        L = curLight->viewPosition.minus(point);
        L.unitize();
        double LdotN = L.dot(trueNormal);
        LdotN = max(0.0, LdotN);
        DoubleColor diffComponent(0.0, 0.0, 0.0, 1.0);
        if (LdotN > 0.0)
            diffComponent.plus(DoubleColor(curLight->diffuse.r*Kd.r*LdotN, curLight->diffuse.g*Kd.g*LdotN, curLight->diffuse.b*Kd.b*LdotN, 1.0));
        shadeColor.plus(diffComponent);

        Float3D Pr = trueNormal.sMult(LdotN);
        Float3D sub = Pr.sMult(2.0);
        R = L.sMult(-1.0).plus(sub);
        R.unitize();
        double RdotV = R.dot(V);
        RdotV = max(0.0, RdotV);
        double cosPhiPower = 0.0;
        if (RdotV > 0.0)
            cosPhiPower = pow(RdotV, theObj->materials[materialIndex].shiny);
        DoubleColor specComponent(curLight->specular.r*Ks.r*cosPhiPower, curLight->specular.g*Ks.g*cosPhiPower, curLight->specular.b*Ks.b*cosPhiPower, 1.0);
        shadeColor.plus(specComponent);
    }
    if (numRecurs >= options->maxRecursiveDepth)
        return shadeColor;*/

    /*if (refractions)
    {
        double rhoNew, rhoOld;
        Float3D norm;
        if (ray.flags == INTERNAL_REFRACT)
        {
            rhoOld = theObj->materials[materialIndex].refractiveIndex;
            rhoNew = rhoAIR;
            norm = Float3D(inv_normal);
        }
        else
        {
            rhoNew = theObj->materials[materialIndex].refractiveIndex;
            rhoOld = rhoAIR;
            norm = Float3D(normal);
        }
        double rhoOldSq = rhoOld * rhoOld;
        double rhoNewSq = rhoNew * rhoNew;
        Float3D d = ray.Rd;
        double dDotn = d.dot(norm);
        Float3D term1 = d.minus(norm.sMult(dDotn)).sMult(rhoOld);
        term1 = term1.sDiv(rhoNew);
        double sqrtOp = 1.0 - ((rhoOldSq*(1.0 - dDotn * dDotn))/rhoNewSq);
        if (sqrtOp < 0.0)
        {
            reflectivity = reflectivity + refractivity;
            reflections = true;
            refractions = false;
        }
        else
        {
            double root = sqrt(sqrtOp);
            Float3D term2 = norm.sMult(root);
            Float3D t = term1.minus(term2);
            t.unitize();
            Ray newRay = Ray(Float3D(), Float3D());
            if (ray.flags == INTERNAL_REFRACT)
                newRay = Ray(t, point, EXTERNAL_REFRACT);
            else
                newRay = Ray(t, point, INTERNAL_REFRACT);
            refrColor = trace(newRay, numRecurs+1);
        }
    }

    if (reflections)
    {
        Float3D Pr = trueNormal.sMult(ray.Rd.dot(trueNormal));
        Float3D sub = Pr.sMult(2.0);
        Float3D refVect = ray.Rd.minus(sub);
        refVect.unitize();

        Ray reflRay = Ray(refVect, point, REFLECT);
        reflColor = trace(reflRay, numRecurs+1);
    }

    DoubleColor rtnColor;
    double shadeWeight;

    if (reflections && !refractions)
    {
        shadeWeight = 1.0 - reflectivity;
        reflColor.scale(reflectivity);
        shadeColor.scale(shadeWeight);
        rtnColor.plus(shadeColor);
        rtnColor.plus(reflColor);
        return rtnColor;
    }
    else if (reflections && refractions)
    {
        shadeWeight = 1.0 - (reflectivity + refractivity);
        reflColor.scale(refractivity);
        reflColor.scale(reflectivity);
        shadeColor.scale(shadeWeight);
        rtnColor.plus(refrColor);
        rtnColor.plus(shadeColor);
        rtnColor.plus(reflColor);
        return rtnColor;
    }
    else if (!reflections && refractions)
    {
        shadeWeight = 1.0 - refractivity;
        reflColor.scale(refractivity);
        shadeColor.scale(shadeWeight);
        rtnColor.plus(refrColor);
        rtnColor.plus(shadeColor);
        return rtnColor;
    }
    else*/
        /*return shadeColor;
}

__device__ bool traceLightRay(Ray ray)
{
    double t = 0.0;
    for (int obj = 0; obj < numObjects; obj++)
    {
        if (intersectSphere(ray, &objects[obj], &t))
        {
            if (abs(t) < 0.0001)
                return false;
            else
                return true;
        }
    }
    return false;
}*/

__device__ bool intersectSphere(Ray *ray, BoundingSphere *theSphere, Float3D viewCenter, float *t)
{
    const float EPS = 0.00001;
    float t0=0.0, t1=0.0, A=0.0, B=0.0, C=0.0, discrim=0.0;
    Float3D RoMinusSc = ray->Ro.minus(viewCenter);
    float fourAC = 0.0;

    A = ray->Rd.dot(ray->Rd);
    B = 2.0 * (ray->Rd.dot(RoMinusSc));
    C = RoMinusSc.dot(RoMinusSc) - theSphere->radiusSq;
    fourAC = (4*A*C);

    discrim = (B*B) - fourAC;

    if (discrim < EPS)
        return false;
    else
    {
        t0 = ((-B) - sqrt(discrim))/(2.0*A);
        t1 = ((-B) + sqrt(discrim))/(2.0*A);

        if (t0 < EPS)
        {
            if (t1 < EPS)
            {
                *t = 0.0;
                return false;
            }
            else
            {
                *t = t1;
                return true;
            }
        }
        else if (t1 < EPS)
        {
            *t = t0;
            return true;
        }
        else if (t0 < t1)
        {
            *t = t0;
            return true;
        }
        else
        {
            *t = t1;
            return true;
        }
    }
}

__device__ bool intersectTriangle(Ray *ray, Mesh *theObj, int v1, int v2, int v3, HitRecord *hrec, bool cull)
{
    Float3D verts[3] = {theObj->vertArray[v1], theObj->vertArray[v2], theObj->vertArray[v3]};
    Float3D edges[2];
    Float3D vnorms[3] = {theObj->viewNormArray[v1], theObj->viewNormArray[v2], theObj->viewNormArray[v3]};
    Float3D pvec, qvec, tvec;
    float det, inv_det;
    float EPSILON = 0.000001;

    edges[0] = verts[1].minus(verts[0]);
    edges[1] = verts[2].minus(verts[0]);
    pvec = ray->Rd.cross(edges[1]);
    det = edges[0].dot(pvec);
    if(cull)
    {
        if (det < EPSILON)
            return false;
        tvec = ray->Ro.minus(verts[0]);
        hrec->u = tvec.dot(pvec);
        if (hrec->u < 0.0 || hrec->u > det)
            return false;
        qvec = tvec.cross(edges[0]);
        hrec->v = ray->Rd.dot(qvec);
        if (hrec->v < 0.0 || hrec->u + hrec->v > det)
            return false;
        hrec->t = edges[1].dot(qvec);
        inv_det = 1.0/det;
        hrec->t *= inv_det;
        hrec->u *= inv_det;
        hrec->v *= inv_det;
    }
    else
    {
        if (det > -EPSILON && det < EPSILON)
            return false;
        inv_det = 1.0/det;
        tvec = ray->Ro.minus(verts[0]);
        hrec->u = tvec.dot(pvec) * inv_det;
        if (hrec->u < 0.0 || hrec->u > 1.0)
            return false;
        qvec = tvec.cross(edges[0]);
        hrec->v = ray->Rd.dot(qvec) * inv_det;
        if (hrec->v < 0.0 || hrec->u + hrec->v > 1.0)
            return false;
        if (det < -EPSILON)
            hrec->backfacing = true;
        else
            hrec->backfacing = false;
        hrec->t = edges[1].dot(qvec) * inv_det;
    }
    if (hrec->t < EPSILON)
        return false;
    else
    {
        hrec->intersectPoint = Float3D((ray->Ro.x + (ray->Rd.x * hrec->t)), (ray->Ro.y + (ray->Rd.y * hrec->t)), (ray->Ro.z + (ray->Rd.z * hrec->t)));
        float w = 1.0 - hrec->u - hrec->v;
        Float3D sumNorms(0.0, 0.0, 0.0);
        vnorms[0] = vnorms[0].sMult(w);
        vnorms[1] = vnorms[1].sMult(hrec->u);
        vnorms[2] = vnorms[2].sMult(hrec->v);
        sumNorms = vnorms[0].plus(vnorms[1].plus(vnorms[2]));
        hrec->normal = sumNorms;
        hrec->normal.unitize();
        return true;
    }
}

void cudaStart(Bitmap *bitmap, Mesh *objects, int numObjects, LightCuda *lights, int numLights, Options *options)
{
    int numRays;
    unsigned char *d_bitmap;
    unsigned char *h_bitmap;
    unsigned char *layers[options->maxRecursiveDepth + 1];
    Mesh *d_objects;
    Mesh *h_objects;
    LightCuda *d_lights;
    Ray *rays;
    Intersect *intersects;

    CHECK_ERROR(hipMalloc((void**)&d_bitmap, bitmap->width * bitmap->height * 3));
    h_bitmap = (unsigned char*)malloc(sizeof(unsigned char) * (bitmap->width * bitmap->height * 3));

    bitmap->data = d_bitmap;

    h_objects = (Mesh *)malloc(sizeof(Mesh) * numObjects);
    memcpy(h_objects, objects, sizeof(Mesh) * numObjects);

    for (int x = 0; x < numObjects; x++)
    {
        h_objects[x].surfaces = new Surface[h_objects[x].numSurfs];
        memcpy(h_objects[x].surfaces, objects[x].surfaces, sizeof(Surface) * objects[x].numSurfs);
        for (int y = 0; y < h_objects[x].numSurfs; y++)
        {
            CHECK_ERROR(hipMalloc((void**)&h_objects[x].surfaces[y].verts, sizeof(int) * h_objects[x].surfaces[y].numVerts));
            CHECK_ERROR(hipMemcpy(h_objects[x].surfaces[y].verts, objects[x].surfaces[y].verts, sizeof(int) * h_objects[x].surfaces[y].numVerts, hipMemcpyHostToDevice));
            delete [] objects[x].surfaces[y].verts;
            objects[x].surfaces[y].verts = h_objects[x].surfaces[y].verts;
            h_objects[x].surfaces[y].verts = NULL;
        }

        delete [] h_objects[x].surfaces;
        CHECK_ERROR(hipMalloc((void **)&h_objects[x].surfaces, sizeof(Surface) * h_objects[x].numSurfs));
        CHECK_ERROR(hipMemcpy(h_objects[x].surfaces, objects[x].surfaces, sizeof(Surface) * h_objects[x].numSurfs, hipMemcpyHostToDevice));
        CHECK_ERROR(hipMalloc((void **)&h_objects[x].materials, sizeof(Material) * h_objects[x].numMats));
        CHECK_ERROR(hipMemcpy(h_objects[x].materials, objects[x].materials, sizeof(Material) * h_objects[x].numMats, hipMemcpyHostToDevice));
        CHECK_ERROR(hipMalloc((void **)&h_objects[x].vertArray, sizeof(Float3D) * h_objects[x].numVerts));
        CHECK_ERROR(hipMemcpy(h_objects[x].vertArray, objects[x].vertArray, sizeof(Float3D) * h_objects[x].numVerts, hipMemcpyHostToDevice));
        CHECK_ERROR(hipMalloc((void **)&h_objects[x].viewNormArray, sizeof(Float3D) * h_objects[x].numVerts));
        CHECK_ERROR(hipMemcpy(h_objects[x].viewNormArray, objects[x].viewNormArray, sizeof(Float3D) * h_objects[x].numVerts, hipMemcpyHostToDevice));
    }

    CHECK_ERROR(hipMalloc((void**)&d_objects, sizeof(Mesh) * numObjects));
    CHECK_ERROR(hipMemcpy(d_objects, h_objects, sizeof(Mesh) * numObjects, hipMemcpyHostToDevice));

    CHECK_ERROR(hipMalloc((void**)&d_lights, sizeof(LightCuda) * numLights));
    CHECK_ERROR(hipMemcpy(d_lights, lights, sizeof(LightCuda) * numLights, hipMemcpyHostToDevice));

    numRays = bitmap->width * bitmap->height;
    CHECK_ERROR(hipMalloc((void**)&rays, sizeof(Ray) * numRays));

    CHECK_ERROR(hipMalloc((void**)&intersects, sizeof(Intersect) * numRays));

    for(int i = 0; i <= options->maxRecursiveDepth; i++)
        CHECK_ERROR(hipMalloc((void**)&layers[i], sizeof(unsigned char) * (bitmap->width * bitmap->height * 4)));

    dim3 blocks((bitmap->width+15)/16, (bitmap->height+15)/16);
    dim3 threads(16, 16);
    //kernel<<<blocks, threads>>>(*bitmap, d_objects, numObjects, d_lights, numLights, *options);
    for(int pass = 0; pass <= options->maxRecursiveDepth; pass++)
    {
        baseKrnl<<<blocks, threads>>>(rays, numRays, *bitmap);
        intersectKrnl<<<blocks, threads>>>(rays, numRays, d_objects, numObjects, options->spheresOnly, intersects, options->cull);
        shadeKrnl<<<blocks, threads>>>(rays, numRays, intersects, layers[pass], d_lights, numLights, *options, pass == options->maxRecursiveDepth ? true : false);
        composeKrnl<<<blocks, threads>>>(*bitmap, layers[pass], pass == options->maxRecursiveDepth ? true : false);
    }

    CHECK_ERROR(hipMemcpy(h_bitmap, d_bitmap, bitmap->width * bitmap->height * 3, hipMemcpyDeviceToHost));

    CHECK_ERROR_FREE(hipFree(d_bitmap), &d_bitmap);

    for (int x = 0; x < numObjects; x++)
    {
        for (int y = 0; y < h_objects[x].numSurfs; y++)
        {
            CHECK_ERROR(hipFree(objects[x].surfaces[y].verts));
            objects[x].surfaces[y].verts = NULL;
        }
        CHECK_ERROR_FREE(hipFree(h_objects[x].surfaces), &h_objects[x].surfaces);
        CHECK_ERROR_FREE(hipFree(h_objects[x].materials), &h_objects[x].materials);
        CHECK_ERROR_FREE(hipFree(h_objects[x].vertArray), &h_objects[x].vertArray);
        CHECK_ERROR_FREE(hipFree(h_objects[x].viewNormArray), &h_objects[x].viewNormArray);
    }
    CHECK_ERROR_FREE(hipFree(d_objects), &d_objects);

    CHECK_ERROR_FREE(hipFree(d_lights), &d_lights);

    CHECK_ERROR_FREE(hipFree(rays), &rays);

    CHECK_ERROR_FREE(hipFree(intersects), &intersects);

    for(int i = 0; i <= options->maxRecursiveDepth; i++)
        CHECK_ERROR_FREE(hipFree(layers[i]), &layers[i]);

    bitmap->data = h_bitmap;

    free(h_objects);
}

void checkError(hipError_t error, const char *file, int line, void **nullObject)
{
    if (nullObject != NULL)
        nullObject = NULL;

    if (error != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(error), file, line);
        exit(EXIT_FAILURE);
    }
}

__global__ void baseKrnl(Ray *rays, int numRays, Bitmap bitmap)
{
    //Map from threadIdx & blockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    if (offset < (bitmap.width * bitmap.height))
    {
        Float3D point(bitmap.firstPixel);
        point.x += (offset % bitmap.width) * bitmap.pixelWidth;
        point.y += ((offset - x) / bitmap.width) * bitmap.pixelHeight;
        rays[offset] = Ray(point.getUnit(), Float3D(0.0, 0.0, 0.0), EYE);
    }
}

__global__ void intersectKrnl(Ray *rays, int numRays, Mesh *objects, int numObjects, bool spheresOnly, Intersect *intrs, bool cull)
{
    //Map from threadIdx & blockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    if (offset < numRays)
    {
        float t = 0.0;
        float intersectDist = 0.0;
        float minDist = 100000000.0;
        int minMatIndex = 0;
        bool minBackfacing = false;
        Mesh *minObj = NULL;
        Float3D minIntPt(0.0, 0.0, 0.0);
        Float3D minNormal(0.0, 0.0, 0.0);
        Float3D intersectPt(0.0, 0.0, 0.0);
        Float3D normal(0.0, 0.0, 0.0);
        Float3D origin(0.0, 0.0, 0.0);

        for (int obj = 0; obj < numObjects; obj++)
        {
            if (intersectSphere(&rays[offset], &(objects[obj].boundingSphere), objects[obj].viewCenter, &t))
            {
                if (abs(t) < 0.0001)
                    continue;
                if (spheresOnly)
                {
                    intersectPt = Float3D((rays[offset].Ro.x+(rays[offset].Rd.x*t)), (rays[offset].Ro.y+(rays[offset].Rd.y*t)), (rays[offset].Ro.z+(rays[offset].Rd.z*t)));
                    normal = (intersectPt.minus(objects[obj].viewCenter).sDiv(objects[obj].boundingSphere.radius));
                    normal.unitize();
                    intersectDist = origin.distanceTo(intersectPt);
                    if (intersectDist < minDist)
                    {
                        minDist = intersectDist;
                        minObj = &objects[obj];
                        minIntPt = Float3D(intersectPt);
                        minNormal = Float3D(normal);
                    }
                }
                else
                {
                    for (int surf = 0; surf < objects[obj].numSurfs; surf++)
                    {
                        for (int i =  0; i < (int)(objects[obj].surfaces[surf].numVerts / 3); i++)
                        {
                            HitRecord hrec;
                            if (intersectTriangle(&rays[offset], &objects[obj], objects[obj].surfaces[surf].verts[i*3], objects[obj].surfaces[surf].verts[(i*3)+1], objects[obj].surfaces[surf].verts[(i*3)+2], &hrec, false))
                            {
                                if (!(rays[offset].flags == EYE && hrec.backfacing && cull) || rays[offset].flags == REFLECT)
                                {
                                    intersectDist = rays[offset].Ro.distanceTo(hrec.intersectPoint);
                                    if (intersectDist < minDist)
                                    {
                                        minDist = intersectDist;
                                        minObj = &objects[obj];
                                        minIntPt = hrec.intersectPoint;
                                        minNormal = hrec.normal;
                                        minMatIndex = objects[obj].surfaces[surf].material;
                                        minBackfacing = hrec.backfacing;
                                    }
                                }
                            }
                        }
                    }
                }
            }
        }
        intrs[offset] = Intersect(minMatIndex, minBackfacing, minObj, minIntPt, minNormal);
    }
}

__global__ void shadeKrnl(Ray *rays, int numRays, Intersect *intrs, unsigned char *layer, LightCuda *lights, int numLights, Options options, bool finalPass)
{
    //Map from threadIdx & blockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    if (offset < numRays && intrs[offset].theObj != NULL)
    {
        int materialIndex = intrs[offset].materialIndex;
        Mesh *theObj = intrs[offset].theObj;
        DoubleColor Ka;
        DoubleColor Kd;
        DoubleColor Ks;
        DoubleColor shadeColor;
        Float3D point = intrs[offset].point;
        Float3D trueNormal(0.0, 0.0, 0.0);
        Float3D inv_normal = intrs[offset].normal.sMult(-1.0);

        layer[offset*4 + 3] = (int) (theObj->materials[materialIndex].reflectivity.r * 255);

        Ka = theObj->materials[materialIndex].ka;
        Kd = theObj->materials[materialIndex].kd;
        Ks = theObj->materials[materialIndex].ks;

        Float3D R(0.0, 0.0, 0.0);
        Float3D L(0.0, 0.0, 0.0);
        Float3D V(0.0, 0.0, 0.0);

        shadeColor.plus(lights[0].ambient);
        V = Float3D(0.0, 0.0, 0.0).minus(point);
        V.unitize();

        if (rays[offset].flags == EYE && intrs[offset].backFacing && !options.cull)
            trueNormal = inv_normal;
        else
            trueNormal = intrs[offset].normal;

        LightCuda *curLight;
        for (int i = 0; i < numLights; i++)
        {
            curLight = &lights[i];

            L = curLight->viewPosition.minus(point);
            L.unitize();
            float LdotN = L.dot(trueNormal);
            LdotN = max(0.0, LdotN);
            DoubleColor diffComponent(0.0, 0.0, 0.0, 1.0);
            if (LdotN > 0.0)
                diffComponent.plus(DoubleColor(curLight->diffuse.r*Kd.r*LdotN, curLight->diffuse.g*Kd.g*LdotN, curLight->diffuse.b*Kd.b*LdotN, 1.0));
            shadeColor.plus(diffComponent);

            Float3D Pr = trueNormal.sMult(LdotN);
            Float3D sub = Pr.sMult(2.0);
            R = L.sMult(-1.0).plus(sub);
            R.unitize();
            float RdotV = R.dot(V);
            RdotV = max(0.0, RdotV);
            float cosPhiPower = 0.0;
            if (RdotV > 0.0)
                cosPhiPower = pow(RdotV, theObj->materials[materialIndex].shiny);
            DoubleColor specComponent(curLight->specular.r*Ks.r*cosPhiPower, curLight->specular.g*Ks.g*cosPhiPower, curLight->specular.b*Ks.b*cosPhiPower, 1.0);
            shadeColor.plus(specComponent);
        }
        if (finalPass)
        {
            layer[offset*4 + 0] = (int) (shadeColor.r * 255);
            layer[offset*4 + 1] = (int) (shadeColor.g * 255);
            layer[offset*4 + 2] = (int) (shadeColor.b * 255);
        }
    }
}

__global__ void composeKrnl(Bitmap bitmap, unsigned char *layer, bool finalPass)
{
    //Map from threadIdx & blockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    if (offset < (bitmap.width * bitmap.height))
    {
        if (finalPass)
        {
            bitmap.data[offset*3 + 0] = layer[offset*4 + 0];
            bitmap.data[offset*3 + 1] = layer[offset*4 + 1];
            bitmap.data[offset*3 + 2] = layer[offset*4 + 2];
        }
        else
        {

        }
    }

}
