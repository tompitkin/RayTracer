#include "hip/hip_runtime.h"
#include "cudaKernel.h"
#include "stdio.h"

__device__ bool intersectSphere(Ray *ray, float radiusSq, Float3D viewCenter, float *t)
{
    const float EPS = 0.00001;
    float t0=0.0, t1=0.0, A=0.0, B=0.0, C=0.0, discrim=0.0;
    Float3D RoMinusSc = ray->Ro.minus(&viewCenter);
    float fourAC = 0.0;

    A = ray->Rd.dot(&ray->Rd);
    B = 2.0 * (ray->Rd.dot(&RoMinusSc));
    C = RoMinusSc.dot(&RoMinusSc) - radiusSq;
    fourAC = (4*A*C);

    discrim = (B*B) - fourAC;

    if (discrim < EPS)
        return false;
    else
    {
        t0 = ((-B) - sqrt(discrim))/(2.0*A);
        t1 = ((-B) + sqrt(discrim))/(2.0*A);

        if (t0 < EPS)
        {
            if (t1 < EPS)
            {
                *t = 0.0;
                return false;
            }
            else
            {
                *t = t1;
                return true;
            }
        }
        else if (t1 < EPS)
        {
            *t = t0;
            return true;
        }
        else if (t0 < t1)
        {
            *t = t0;
            return true;
        }
        else
        {
            *t = t1;
            return true;
        }
    }
}

__device__ bool intersectTriangle(Ray *ray, Float3D *tri, float *t, bool *backFacing, float *v, float *u)
{
    Float3D pvec, qvec, tvec;
    float det, inv_det;
    const float EPSILON = 0.00001;

    pvec = ray->Rd.cross(&tri[2]);
    det = tri[1].dot(&pvec);

    if (det > -EPSILON && det < EPSILON)
        return false;
    inv_det = 1.0/det;
    tvec = ray->Ro.minus(&tri[0]);
    *u = tvec.dot(&pvec) * inv_det;
    if (*u < 0.0 || *u > 1.0)
        return false;
    qvec = tvec.cross(&tri[1]);
    *v = ray->Rd.dot(&qvec) * inv_det;
    if (*v < 0.0 || *u + *v > 1.0)
        return false;
    if (det < -EPSILON)
        *backFacing = true;
    else
        *backFacing = false;
    *t = tri[2].dot(&qvec) * inv_det;

    if (*t < EPSILON)
        return false;

    return true;
}

void cudaStart(Bitmap *bitmap, Mesh *objects, int numObjects, LightCuda *lights, int numLights, Options *options)
{
    int numRays;
    unsigned char *d_bitmap;
    unsigned char *h_bitmap;
    unsigned char *layers[options->maxRecursiveDepth + 1];
    bool *hits;
    Mesh *d_objects;
    Mesh *h_objects;
    LightCuda *d_lights;
    Ray *rays;
    Intersect *intersects;

    hipEvent_t start, stop;
    CHECK_ERROR(hipEventCreate(&start));
    CHECK_ERROR(hipEventCreate(&stop));
    CHECK_ERROR(hipEventRecord(start, 0));

    CHECK_ERROR(hipMalloc((void**)&d_bitmap, bitmap->width * bitmap->height * 3));
    h_bitmap = (unsigned char*)malloc(sizeof(unsigned char) * (bitmap->width * bitmap->height * 3));

    bitmap->data = d_bitmap;

    h_objects = (Mesh *)malloc(sizeof(Mesh) * numObjects);
    memcpy(h_objects, objects, sizeof(Mesh) * numObjects);

    for (int x = 0; x < numObjects; x++)
    {
        h_objects[x].surfaces = new Surface[h_objects[x].numSurfs];
        memcpy(h_objects[x].surfaces, objects[x].surfaces, sizeof(Surface) * objects[x].numSurfs);
        for (int y = 0; y < h_objects[x].numSurfs; y++)
        {
            CHECK_ERROR(hipMalloc((void**)&h_objects[x].surfaces[y].vertArray, sizeof(Float3D) * h_objects[x].surfaces[y].numVerts));
            CHECK_ERROR(hipMemcpy(h_objects[x].surfaces[y].vertArray, objects[x].surfaces[y].vertArray, sizeof(Float3D) * h_objects[x].surfaces[y].numVerts, hipMemcpyHostToDevice));
            CHECK_ERROR(hipMalloc((void**)&h_objects[x].surfaces[y].viewNormArray, sizeof(Float3D) * h_objects[x].surfaces[y].numVerts));
            CHECK_ERROR(hipMemcpy(h_objects[x].surfaces[y].viewNormArray, objects[x].surfaces[y].viewNormArray, sizeof(Float3D) * h_objects[x].surfaces[y].numVerts, hipMemcpyHostToDevice));
            delete [] objects[x].surfaces[y].vertArray;
            delete [] objects[x].surfaces[y].viewNormArray;
            objects[x].surfaces[y].vertArray = h_objects[x].surfaces[y].vertArray;
            objects[x].surfaces[y].viewNormArray = h_objects[x].surfaces[y].viewNormArray;
            h_objects[x].surfaces[y].vertArray = NULL;
            h_objects[x].surfaces[y].viewNormArray = NULL;
        }

        delete [] h_objects[x].surfaces;
        CHECK_ERROR(hipMalloc((void **)&h_objects[x].surfaces, sizeof(Surface) * h_objects[x].numSurfs));
        CHECK_ERROR(hipMemcpy(h_objects[x].surfaces, objects[x].surfaces, sizeof(Surface) * h_objects[x].numSurfs, hipMemcpyHostToDevice));
        CHECK_ERROR(hipMalloc((void **)&h_objects[x].materials, sizeof(Material) * h_objects[x].numMats));
        CHECK_ERROR(hipMemcpy(h_objects[x].materials, objects[x].materials, sizeof(Material) * h_objects[x].numMats, hipMemcpyHostToDevice));
    }

    CHECK_ERROR(hipMalloc((void**)&d_objects, sizeof(Mesh) * numObjects));
    CHECK_ERROR(hipMemcpy(d_objects, h_objects, sizeof(Mesh) * numObjects, hipMemcpyHostToDevice));

    CHECK_ERROR(hipMalloc((void**)&d_lights, sizeof(LightCuda) * numLights));
    CHECK_ERROR(hipMemcpy(d_lights, lights, sizeof(LightCuda) * numLights, hipMemcpyHostToDevice));

    numRays = bitmap->width * bitmap->height;
    CHECK_ERROR(hipMalloc((void**)&rays, sizeof(Ray) * numRays));

    CHECK_ERROR(hipMalloc((void**)&intersects, sizeof(Intersect) * numRays));

    CHECK_ERROR(hipMalloc((void**)&hits, sizeof(bool) * numRays));

    for(int i = 0; i <= options->maxRecursiveDepth; i++)
    {
        CHECK_ERROR(hipMalloc((void**)&layers[i], sizeof(unsigned char) * (bitmap->width * bitmap->height * 4)));
        CHECK_ERROR(hipMemset(layers[i], 0, sizeof(unsigned char) * bitmap->width * bitmap->height * 4));
    }

    dim3 blocks((bitmap->width+15)/16, (bitmap->height+15)/16);
    dim3 threads(16, 16);
    baseKrnl<<<blocks, threads>>>(rays, *bitmap);
    for(int pass = 0; pass <= options->maxRecursiveDepth; pass++)
    {
        CHECK_ERROR(hipMemset(hits, false, sizeof(bool) * numRays));
        initIntersectKrnl<<<blocks, threads>>>(numRays, intersects);
        intersectSphereKrnl<<<blocks, threads>>>(rays, numRays, d_objects, numObjects, options->spheresOnly, intersects, hits);
        if (!options->spheresOnly)
        {
            for (int obj = 0; obj < numObjects; obj++)
            {
                for (int surf = 0; surf < objects[obj].numSurfs; surf++)
                {
                    for (int offset = 0; offset < (int)ceil(((float)objects[obj].surfaces[surf].numVerts) / CHUNK); offset++)
                    {
                        intersectTriangleKrnl<<<blocks, threads>>>(rays, numRays, intersects, hits, &d_objects[obj], &objects[obj].surfaces[surf].vertArray[offset * CHUNK], &objects[obj].surfaces[surf].viewNormArray[offset * CHUNK], (objects[obj].surfaces[surf].numVerts - offset * CHUNK) < CHUNK ? (objects[obj].surfaces[surf].numVerts - offset * CHUNK) : CHUNK , objects[obj].surfaces[surf].material);
                    }
                }
            }
        }
        shadeKrnl<<<blocks, threads>>>(rays, numRays, intersects, layers[pass], d_lights, numLights, *options);
        if (options->reflections)
            reflectKrnl<<<blocks, threads>>>(rays, numRays, intersects);
    }
    for(int pass = options->maxRecursiveDepth; pass >= 0; pass--)
        composeKrnl<<<blocks, threads>>>(*bitmap, layers[pass], pass == options->maxRecursiveDepth ? true : false);

    CHECK_ERROR(hipMemcpy(h_bitmap, d_bitmap, bitmap->width * bitmap->height * 3, hipMemcpyDeviceToHost));

    CHECK_ERROR(hipEventRecord(stop, 0));
    CHECK_ERROR(hipEventSynchronize(stop));

    float time;
    CHECK_ERROR(hipEventElapsedTime(&time, start, stop));

    printf("(CUDA) Ray Trace total time: %3.1f ms\n", time);

    CHECK_ERROR(hipEventDestroy(start));
    CHECK_ERROR(hipEventDestroy(stop));

    CHECK_ERROR_FREE(hipFree(d_bitmap), &d_bitmap);

    for (int x = 0; x < numObjects; x++)
    {
        for (int y = 0; y < h_objects[x].numSurfs; y++)
        {
            CHECK_ERROR(hipFree(objects[x].surfaces[y].vertArray));
            CHECK_ERROR(hipFree(objects[x].surfaces[y].viewNormArray));
            objects[x].surfaces[y].vertArray = NULL;
            objects[x].surfaces[y].viewNormArray = NULL;
        }
        CHECK_ERROR_FREE(hipFree(h_objects[x].surfaces), &h_objects[x].surfaces);
        CHECK_ERROR_FREE(hipFree(h_objects[x].materials), &h_objects[x].materials);
    }
    CHECK_ERROR_FREE(hipFree(d_objects), &d_objects);

    CHECK_ERROR_FREE(hipFree(d_lights), &d_lights);

    CHECK_ERROR_FREE(hipFree(rays), &rays);

    CHECK_ERROR_FREE(hipFree(intersects), &intersects);

    CHECK_ERROR_FREE(hipFree(hits), &hits);

    for(int i = 0; i <= options->maxRecursiveDepth; i++)
        CHECK_ERROR_FREE(hipFree(layers[i]), &layers[i]);

    bitmap->data = h_bitmap;

    free(h_objects);
}

void checkError(hipError_t error, const char *file, int line, void **nullObject)
{
    if (nullObject != NULL)
        nullObject = NULL;

    if (error != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(error), file, line);
        exit(EXIT_FAILURE);
    }
}

__global__ void baseKrnl(Ray *rays, Bitmap bitmap)
{
    //Map from threadIdx & blockIdx to pixel position
    int offset = (threadIdx.x + blockIdx.x * blockDim.x) + (threadIdx.y + blockIdx.y * blockDim.y) * blockDim.x * gridDim.x;

    if (offset < (bitmap.width * bitmap.height))
    {
        Float3D point(bitmap.firstPixel);
        point.x += (offset % bitmap.width) * bitmap.pixelWidth;
        point.y += ((offset - (threadIdx.x + blockIdx.x * blockDim.x)) / bitmap.width) * bitmap.pixelHeight;
        rays[offset].Rd = point.getUnit();
        rays[offset].Ro = Float3D(0.0, 0.0, 0.0);
        rays[offset].flags = EYE;
    }
}

__global__ void initIntersectKrnl(int numIntrs, Intersect *intrs)
{
    int offset = (threadIdx.x + blockIdx.x * blockDim.x) + (threadIdx.y + blockIdx.y * blockDim.y) * blockDim.x * gridDim.x;

    if (offset < numIntrs)
    {
        intrs[offset].theObj = NULL;
        intrs[offset].distance = 100000000.0;
    }
}

__global__ void intersectSphereKrnl(Ray *rays, int numRays, Mesh *objects, int numObjects, bool spheresOnly, Intersect *intrs, bool *hits)
{
    //Map from threadIdx & blockIdx to pixel position
    int offset = (threadIdx.x + blockIdx.x * blockDim.x) + (threadIdx.y + blockIdx.y * blockDim.y) * blockDim.x * gridDim.x;

    __shared__ Ray R[16][16];

    if (offset < numRays)
    {
        R[threadIdx.y][threadIdx.x] = rays[offset];

        float t = 0.0;
        float intersectDist = 0.0;
        float minDist = 100000000.0;
        Float3D intersectPt;
        Float3D normal;
        Mesh *theObj;

        for (int obj = 0; obj < numObjects; obj++)
        {
            theObj = &objects[obj];
            if (intersectSphere(&R[threadIdx.y][threadIdx.x], theObj->boundingSphere.radiusSq, theObj->viewCenter, &t))
            {
                if (abs(t) < 0.0001)
                    continue;
                if (spheresOnly)
                {
                    intersectPt = Float3D((R[threadIdx.y][threadIdx.x].Ro.x+(R[threadIdx.y][threadIdx.x].Rd.x*t)), (R[threadIdx.y][threadIdx.x].Ro.y+(R[threadIdx.y][threadIdx.x].Rd.y*t)), (R[threadIdx.y][threadIdx.x].Ro.z+(R[threadIdx.y][threadIdx.x].Rd.z*t)));
                    normal = (intersectPt.minus(&theObj->viewCenter).sDiv(theObj->boundingSphere.radius));
                    normal.unitize();
                    intersectDist = Float3D(0.0, 0.0, 0.0).distanceTo(&intersectPt);
                    if (intersectDist < minDist)
                    {
                        minDist = intersectDist;
                        intrs[offset] = Intersect(0, false, theObj, intersectPt, normal, minDist);
                    }
                }
                else
                {
                    hits[offset] = true;
                }
            }
        }
    }
}

__global__ void intersectTriangleKrnl(Ray *rays, int numRays, Intersect *intrs, bool *hits, Mesh *theObj, Float3D *verts, Float3D *norms, int numVerts, int mat)
{
    int offset = (threadIdx.x + blockIdx.x * blockDim.x) + (threadIdx.y + blockIdx.y * blockDim.y) * blockDim.x * gridDim.x;

    __shared__ Float3D V[CHUNK];
    __shared__ Float3D N[CHUNK];
    __shared__ Ray R[16][16];

    if (offset < numRays)
    {
        R[threadIdx.y][threadIdx.x] = rays[offset];
        float t, v, u, w;
        bool backFacing;
        float intersectDist = 0.0;
        float minDist = intrs[offset].distance;
        int index = (threadIdx.y + threadIdx.x * 16) % numVerts;

        V[index] = verts[index];
        N[index] = norms[index];

        __syncthreads();

        if (hits[offset])
        {
            for (int i =  0; i < (numVerts / 3); i++)
            {
                if (intersectTriangle(&R[threadIdx.y][threadIdx.x], &V[i * 3], &t, &backFacing, &v, &u))
                {
                    Float3D intersectPt = Float3D((R[threadIdx.y][threadIdx.x].Ro.x+(R[threadIdx.y][threadIdx.x].Rd.x*t)), (R[threadIdx.y][threadIdx.x].Ro.y+(R[threadIdx.y][threadIdx.x].Rd.y*t)), (R[threadIdx.y][threadIdx.x].Ro.z+(R[threadIdx.y][threadIdx.x].Rd.z*t)));
                    intersectDist = R[threadIdx.y][threadIdx.x].Ro.distanceTo(&intersectPt);
                    if (intersectDist < minDist)
                    {
                        w = 1.0 - u - v;
                        Float3D normal = N[i * 3 + 2].sMult(v);
                        normal = N[i * 3 + 1].sMult(u).plus(&normal);
                        normal = N[i * 3].sMult(w).plus(&normal);
                        normal.unitize();
                        minDist = intersectDist;
                        intrs[offset] = Intersect(mat, backFacing, theObj, intersectPt, normal, minDist);
                    }
                }
            }
        }
    }
}

__global__ void shadeKrnl(Ray *rays, int numRays, Intersect *intrs, unsigned char *layer, LightCuda *lights, int numLights, Options options)
{
    //Map from threadIdx & blockIdx to pixel position
    int offset = (threadIdx.x + blockIdx.x * blockDim.x) + (threadIdx.y + blockIdx.y * blockDim.y) * blockDim.x * gridDim.x;

    if (offset < numRays && intrs[offset].theObj != NULL)
    {
        int materialIndex = intrs[offset].materialIndex;
        Mesh *theObj = intrs[offset].theObj;
        FloatColor Ka(0.0, 0.0, 0.0, 1.0);
        FloatColor Kd(0.0, 0.0, 0.0, 1.0);
        FloatColor Ks(0.0, 0.0, 0.0, 1.0);
        FloatColor shadeColor(0.0, 0.0, 0.0, 1.0);
        FloatColor ambColor(0.0, 0.0, 0.0, 1.0);
        Float3D point = intrs[offset].point;
        Float3D trueNormal(0.0, 0.0, 0.0);
        Float3D inv_normal = intrs[offset].normal.sMult(-1.0);
        Float3D R(0.0, 0.0, 0.0);
        Float3D L(0.0, 0.0, 0.0);
        Float3D V(0.0, 0.0, 0.0);

        layer[offset*4 + 3] = (int) (theObj->materials[materialIndex].reflectivity.r * 255);

        Ka = theObj->materials[materialIndex].ka;
        Kd = theObj->materials[materialIndex].kd;
        Ks = theObj->materials[materialIndex].ks;

        ambColor.r = Ka.r * lights[0].ambient.r;
        ambColor.g = Ka.g * lights[0].ambient.g;
        ambColor.b = Ka.b * lights[0].ambient.b;

        shadeColor.plus(ambColor);
        V = Float3D(0.0, 0.0, 0.0).minus(&point);
        V.unitize();

        if (rays[offset].flags == EYE && intrs[offset].backFacing)
            trueNormal = inv_normal;
        else
            trueNormal = intrs[offset].normal;

        LightCuda *curLight;
        for (int i = 0; i < numLights; i++)
        {
            curLight = &lights[i];

            L = curLight->viewPosition.minus(&point);
            L.unitize();
            float LdotN = L.dot(&trueNormal);
            LdotN = max(0.0, LdotN);
            FloatColor diffComponent(0.0, 0.0, 0.0, 1.0);
            if (LdotN > 0.0)
                diffComponent.plus(FloatColor(curLight->diffuse.r*Kd.r*LdotN, curLight->diffuse.g*Kd.g*LdotN, curLight->diffuse.b*Kd.b*LdotN, 1.0));
            shadeColor.plus(diffComponent);

            Float3D Pr = trueNormal.sMult(LdotN);
            Float3D sub = Pr.sMult(2.0);
            R = L.sMult(-1.0).plus(&sub);
            R.unitize();
            float RdotV = R.dot(&V);
            RdotV = max(0.0, RdotV);
            float cosPhiPower = 0.0;
            if (RdotV > 0.0)
                cosPhiPower = pow(RdotV, theObj->materials[materialIndex].shiny);
            FloatColor specComponent(curLight->specular.r*Ks.r*cosPhiPower, curLight->specular.g*Ks.g*cosPhiPower, curLight->specular.b*Ks.b*cosPhiPower, 1.0);
            shadeColor.plus(specComponent);
        }

        layer[offset*4 + 0] = (shadeColor.r < 0.0 ? 0.0 : (shadeColor.r > 1.0 ? 1.0 : shadeColor.r)) * 255.0;
        layer[offset*4 + 1] = (shadeColor.g < 0.0 ? 0.0 : (shadeColor.g > 1.0 ? 1.0 : shadeColor.g)) * 255.0;
        layer[offset*4 + 2] = (shadeColor.b < 0.0 ? 0.0 : (shadeColor.b > 1.0 ? 1.0 : shadeColor.b)) * 255.0;
    }
}

__global__ void reflectKrnl(Ray *rays, int numRays, Intersect *intrs)
{
    int offset = (threadIdx.x + blockIdx.x * blockDim.x) + (threadIdx.y + blockIdx.y * blockDim.y) * blockDim.x * gridDim.x;

    if (offset < numRays && intrs[offset].theObj != NULL)
    {
        Float3D trueNormal;

        if (rays[offset].flags == EYE && intrs[offset].backFacing)
            trueNormal = intrs[offset].normal.sMult(-1.0);
        else
            trueNormal = intrs[offset].normal;

        Float3D Pr = trueNormal.sMult(rays[offset].Rd.dot(&trueNormal));
        Float3D sub = Pr.sMult(2.0);
        Float3D refVect = rays[offset].Rd.minus(&sub);
        refVect.unitize();

        rays[offset] = Ray(refVect, intrs[offset].point, REFLECT);
    }
}

__global__ void composeKrnl(Bitmap bitmap, unsigned char *layer, bool finalPass)
{
    //Map from threadIdx & blockIdx to pixel position
    int offset = (threadIdx.x + blockIdx.x * blockDim.x) + (threadIdx.y + blockIdx.y * blockDim.y) * blockDim.x * gridDim.x;

    if (offset < (bitmap.width * bitmap.height))
    {
        if (finalPass)
        {
            bitmap.data[offset*3 + 0] = layer[offset*4 + 0];
            bitmap.data[offset*3 + 1] = layer[offset*4 + 1];
            bitmap.data[offset*3 + 2] = layer[offset*4 + 2];
        }
        else
        {
            FloatColor shadeColor(layer[offset*4 + 0]/255.0, layer[offset*4 + 1]/255.0, layer[offset*4 + 2]/255.0, 1.0);
            FloatColor reflColor(bitmap.data[offset*3 + 0]/255.0, bitmap.data[offset*3 + 1]/255.0, bitmap.data[offset*3 + 2]/255.0, 1.0);
            shadeColor.scale(1.0 - (float)(layer[offset*4 + 3] / 255.0));
            reflColor.scale((float)layer[offset*4 + 3] / 255.0);
            shadeColor.plus(reflColor);
            bitmap.data[offset*3 + 0] = (shadeColor.r < 0.0 ? 0.0 : (shadeColor.r > 1.0 ? 1.0 : shadeColor.r)) * 255.0;
            bitmap.data[offset*3 + 1] = (shadeColor.g < 0.0 ? 0.0 : (shadeColor.g > 1.0 ? 1.0 : shadeColor.g)) * 255.0;
            bitmap.data[offset*3 + 2] = (shadeColor.b < 0.0 ? 0.0 : (shadeColor.b > 1.0 ? 1.0 : shadeColor.b)) * 255.0;
        }
    }

}
